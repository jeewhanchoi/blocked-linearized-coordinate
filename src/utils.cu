#include "hip/hip_runtime.h"
#include "common.hpp"
#include "utils.hpp"
#include <iomanip>
#include <cmath>
#include <cassert>


#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif


template <typename T> T* make_device_copy(T* vector, IType n, std::string name) {
    T* d_vector = nullptr;
    check_cuda(hipMalloc(&d_vector, sizeof(T) * n), "hipMalloc " + name);
    check_cuda(hipMemcpy(d_vector, vector, sizeof(T) * n, hipMemcpyHostToDevice), "hipMemcpy " + name);
    return d_vector;
}
template IType* make_device_copy(IType* vector, IType n, std::string name);
template FType* make_device_copy(FType* vector, IType n, std::string name);
template FType** make_device_copy(FType** vector, IType n, std::string name);
template FType*** make_device_copy(FType*** vector, IType n, std::string name);
template unsigned long* make_device_copy(unsigned long* vector, IType n, std::string name);
template int* make_device_copy(int* vector, IType n, std::string name);


template <typename T> bool vectors_equal(T* x_cpu, T* x_gpu, IType n) {
    T* x_gpu_host = new T[n];
    assert(x_gpu_host);
    check_cuda(
        hipMemcpy(x_gpu_host, x_gpu, sizeof(T) * n, hipMemcpyDeviceToHost),
        "hipMemcpy"
    );

    bool equal = 1;
    for (IType i = 0; i < n; i++) {
        if (x_cpu[i] != x_gpu_host[i]) {
            std::cerr << "Warning: index " << i << " failed to match" << std::endl;
            std::cerr << "CPU " << x_cpu[i] << " vs GPU " << x_gpu_host[i] << std::endl;
            equal = 0;
            break;
        }
    }
    
    delete [] x_gpu_host;
    return equal;
}
template bool vectors_equal(IType* x_cpu, IType* x_gpu, IType n);


template <typename T> bool vectors_equal(T* x_cpu, T* x_gpu, IType n, FType tolerance) {
    T* x_gpu_host = new T[n];
    assert(x_gpu_host);
    check_cuda(
        hipMemcpy(x_gpu_host, x_gpu, sizeof(T) * n, hipMemcpyDeviceToHost),
        "hipMemcpy"
    );

    //print_vector(x_cpu, n, "CPU");
    //print_vector(x_gpu_host, n, "GPU");

    bool equal = 1;
    FType tol;
    for (IType i = 0; i < n; i++) {
        tol = max(abs(x_cpu[i]) * tolerance, tolerance);
        if (abs(x_cpu[i] - x_gpu_host[i]) > tol) {
            std::cerr << "Warning: index " << i << " failed to match" << std::endl;
            std::cerr << "CPU " << std::setprecision(20) << x_cpu[i] << " vs GPU " << std::setprecision(20) << x_gpu_host[i];
            std::cerr << ", tolerance " << tol << ", difference " << abs(x_cpu[i] - x_gpu_host[i]) << std::endl;
            equal = 0;
            break;
        }
    }
    
    delete [] x_gpu_host;
    return equal;
}
template bool vectors_equal(FType* x_cpu, FType* x_gpu, IType n, FType tolerance);


template <typename T> void print_vector(T* x, IType n, std::string name) {
    std::cout << "\"" << name << "\": Vector with " << n << " elements" << std::endl;
    std::cout << " ";
    for (IType i = 0; i < n; i++) std::cout << " " << x[i];
    std::cout << std::endl;
}
template void print_vector(FType* x, IType n, std::string name);
template void print_vector(IType* x, IType n, std::string name);


template <typename T> void print_matrix_col_maj(T* mtx, IType m, IType n, std::string name) {
    std::cout << "\"" << name << "\": " << m << " x " << n << " matrix" << std::endl;
    for (IType i = 0; i < m; i++) {
        std::cout << " ";
        for (IType j = 0; j < n; j++) {
            std::cout << " " << mtx[j * m + i];
        }
        std::cout << std::endl;
    }
}
template void print_matrix_col_maj(FType* mtx, IType m, IType n, std::string name);
template void print_matrix_col_maj(IType* mtx, IType m, IType n, std::string name);


template <typename T> void print_matrix_row_maj(T* mtx, IType m, IType n, std::string name) {
    std::cout << "\"" << name << "\": " << m << " x " << n << " matrix" << std::endl;
    for (IType i = 0; i < m; i++) {
        std::cout << " ";
        for (IType j = 0; j < n; j++) {
            std::cout << " " << mtx[i * m + j];
        }
        std::cout << std::endl;
    }
}
template void print_matrix_row_maj(FType* mtx, IType m, IType n, std::string name);
template void print_matrix_row_maj(IType* mtx, IType m, IType n, std::string name);


void check_cuda(hipError_t status, std::string message) {
    if (status != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(status);
        std::cerr << ". " << message << std::endl;
        exit(EXIT_FAILURE);
    }
}


std::string cublasGetErrorString(hipblasStatus_t status) {
    // Source: https://stackoverflow.com/questions/13041399/equivalent-of-cudageterrorstring-for-cublas
    switch(status) {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}
void check_cublas(hipblasStatus_t status, std::string message) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "Error: " << cublasGetErrorString(status);
        std::cerr << ". " << message << std::endl;
        exit(EXIT_FAILURE);
    }
}


std::string cusolverGetErrorString(hipsolverStatus_t status) {
    switch(status) {
        case HIPSOLVER_STATUS_SUCCESS: return "HIPSOLVER_STATUS_SUCCESS";
        case HIPSOLVER_STATUS_NOT_INITIALIZED: return "HIPSOLVER_STATUS_NOT_INITIALIZED";
        case HIPSOLVER_STATUS_ALLOC_FAILED: return "HIPSOLVER_STATUS_ALLOC_FAILED";
        case HIPSOLVER_STATUS_INVALID_VALUE: return "HIPSOLVER_STATUS_INVALID_VALUE";
        case HIPSOLVER_STATUS_ARCH_MISMATCH: return "HIPSOLVER_STATUS_ARCH_MISMATCH";
        case HIPSOLVER_STATUS_EXECUTION_FAILED: return "HIPSOLVER_STATUS_EXECUTION_FAILED";
        case HIPSOLVER_STATUS_INTERNAL_ERROR: return "HIPSOLVER_STATUS_INTERNAL_ERROR";
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    }
    return "unknown error";
}
void check_cusolver(hipsolverStatus_t status, std::string message) {
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "Error: " << cusolverGetErrorString(status);
        std::cerr << ". " << message << std::endl;
        exit(EXIT_FAILURE);
    }
}
